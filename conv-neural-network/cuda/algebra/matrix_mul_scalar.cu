#include "hip/hip_runtime.h"
#include "matrix_mul_scalar.cuh"

#define BLOCK_DIM 32

namespace cuda {

	template <class _Ty>
	__global__ void matrix_mul_scalar_kernel(const _Ty* A, _Ty* B, _Ty C, size_t N, size_t M) {
		int i = blockIdx.y * blockDim.y + threadIdx.y;
		int j = blockIdx.x * blockDim.x + threadIdx.x;

		if (i < M && j < N) {
			B[i * N + j] = A[i * N + j] * C;
		}
	}

	template <class _Ty>
	void _matrix_mul_scalar(const _Ty* A, _Ty* B, _Ty _Val, size_t N, size_t M) {
		const dim3 threads(BLOCK_DIM, BLOCK_DIM);
		const dim3 blocks((N - 1) / BLOCK_DIM + 1, (M - 1) / BLOCK_DIM + 1);

		matrix_mul_scalar_kernel<<<blocks, threads>>>(A, B, _Val, N, M);
	}

	template void _matrix_mul_scalar<bfloat16>(const bfloat16*, bfloat16*, bfloat16, size_t, size_t);
	template void _matrix_mul_scalar<float>(const float*, float*, float, size_t, size_t);
	template void _matrix_mul_scalar<double>(const double*, double*, double, size_t, size_t);

}