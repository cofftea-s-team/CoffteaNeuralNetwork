#include "hip/hip_runtime.h"
#include "matrix_scalar_mul.cuh"

#define BLOCK_DIM 32

namespace cuda {

	template <bool _T1, bool _T2, class _Ty>
	__global__ void matrix_scalar_mul_kernel(const _Ty* A, const _Ty* B, _Ty* C, size_t N, size_t M) {
		int i = blockIdx.y * BLOCK_DIM + threadIdx.y;
		int j = blockIdx.x * BLOCK_DIM + threadIdx.x;

		if (i < N && j < M) {
			if constexpr (_T1 && _T2) {
				C[i * M + j] = A[j * N + i] * B[j * N + i];
			}
			else if constexpr (_T1 && !_T2) {
				C[i * M + j] = A[j * N + i] * B[i * M + j];
			}
			else if constexpr (!_T1 && _T2) {
				C[i * M + j] = A[i * M + j] * B[j * N + i];
			}
			else {
				C[i * M + j] = A[i * M + j] * B[i * M + j];
			}
		}
	}

	template <bool _T1, bool _T2, class _Ty>
	void _matrix_scalar_mul(const _Ty* A, const _Ty* B, _Ty* C, size_t N, size_t M) {
		const dim3 threads(BLOCK_DIM, BLOCK_DIM);
		const dim3 blocks((N - 1) / BLOCK_DIM + 1, (M - 1) / BLOCK_DIM + 1);

		matrix_scalar_mul_kernel<_T1, _T2>
			<< <blocks, threads >> > (A, B, C, N, M);
	}

	template void _matrix_scalar_mul<false, false, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);
	template void _matrix_scalar_mul<true, false, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);
	template void _matrix_scalar_mul<false, true, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);
	template void _matrix_scalar_mul<true, true, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);

	template void _matrix_scalar_mul<false, false, float>(const float*, const float*, float*, size_t, size_t);
	template void _matrix_scalar_mul<true, false, float>(const float*, const float*, float*, size_t, size_t);
	template void _matrix_scalar_mul<false, true, float>(const float*, const float*, float*, size_t, size_t);
	template void _matrix_scalar_mul<true, true, float>(const float*, const float*, float*, size_t, size_t);

	template void _matrix_scalar_mul<false, false, double>(const double*, const double*, double*, size_t, size_t);
	template void _matrix_scalar_mul<true, false, double>(const double*, const double*, double*, size_t, size_t);
	template void _matrix_scalar_mul<false, true, double>(const double*, const double*, double*, size_t, size_t);
	template void _matrix_scalar_mul<true, true, double>(const double*, const double*, double*, size_t, size_t);
}