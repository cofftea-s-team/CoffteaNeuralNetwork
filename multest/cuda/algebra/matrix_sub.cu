#include "hip/hip_runtime.h"
#include "matrix_sub.cuh"

namespace cuda {
	template <bool _T1, bool _T2, class _Ty>
	__global__ void matrix_sub_kernel(const _Ty* A, const _Ty* B, _Ty* C, size_t N, size_t M) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;

		if (i < N && j < M) {
			if constexpr (_T1 && _T2) {
				C[i * M + j] = A[j * N + i] - B[j * N + i];
			}
			else if constexpr (_T1 && !_T2) {
				C[i * M + j] = A[j * N + i] - B[i * M + j];
			}
			else if constexpr (!_T1 && _T2) {
				C[i * M + j] = A[i * M + j] - B[j * N + i];
			}
			else {
				C[i * M + j] = A[i * M + j] - B[i * M + j];
			}
		}
	}
	
	template <bool _T1, bool _T2, class _Ty>
	void _matrix_sub(const _Ty* A, const _Ty* B, _Ty* C, size_t N, size_t M) {
		const dim3 blockDim(32, 32);
		const dim3 gridDim((N - 1) / 32 + 1, (M - 1) / 32 + 1);

		matrix_sub_kernel<_T1, _T2>
			<<<gridDim, blockDim>>>(A, B, C, N, M);
	}

	template void _matrix_sub<false, false, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);
	template void _matrix_sub<true, false, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);
	template void _matrix_sub<false, true, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);
	template void _matrix_sub<true, true, bfloat16>(const bfloat16*, const bfloat16*, bfloat16*, size_t, size_t);

	template void _matrix_sub<false, false, float>(const float*, const float*, float*, size_t, size_t);
	template void _matrix_sub<true, false, float>(const float*, const float*, float*, size_t, size_t);
	template void _matrix_sub<false, true, float>(const float*, const float*, float*, size_t, size_t);
	template void _matrix_sub<true, true, float>(const float*, const float*, float*, size_t, size_t);

	template void _matrix_sub<false, false, double>(const double*, const double*, double*, size_t, size_t);
	template void _matrix_sub<true, false, double>(const double*, const double*, double*, size_t, size_t);
	template void _matrix_sub<false, true, double>(const double*, const double*, double*, size_t, size_t);
	template void _matrix_sub<true, true, double>(const double*, const double*, double*, size_t, size_t);
}