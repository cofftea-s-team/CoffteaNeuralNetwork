#include "hip/hip_runtime.h"
#include "matrix_transpose.cuh"


namespace cuda {

    template <class _Ty>
    __global__ void transpose_kernel(const _Ty* _Src, _Ty* _Dst, size_t _Rows, size_t _Cols)
    {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;
		
		if (i < _Rows && j < _Cols)
			_Dst[i * _Cols + j] = _Src[j * _Rows + i];
    }

	template<class _Ty>
	void _matrix_transpose(const _Ty* _Src, _Ty* _Dst, size_t _Rows, size_t _Cols)
	{
        const dim3 threadsPerBlock(32, 32);
        const dim3 numBlocks((_Rows + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (_Cols + threadsPerBlock.y - 1) / threadsPerBlock.y);

        transpose_kernel<<<numBlocks, threadsPerBlock>>>(_Src, _Dst, _Rows, _Cols);
	}
	
	template void _matrix_transpose(const double* _Src, double* _Dst, size_t _Rows, size_t _Cols);
	template void _matrix_transpose(const float* _Src, float* _Dst, size_t _Rows, size_t _Cols);
	template void _matrix_transpose(const bfloat16* _Src, bfloat16* _Dst, size_t _Rows, size_t _Cols);
}